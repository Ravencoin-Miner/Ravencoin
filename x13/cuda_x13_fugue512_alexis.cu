#include "hip/hip_runtime.h"
/*
 * Quick and dirty addition of Fugue-512 for X13
 *
 * Built on cbuchner1's implementation, actual hashing code
 * heavily based on phm's sgminer
 *
 *
 */
#include "cuda_helper_alexis.h"
#include "miner.h"
#include "cuda_vectors_alexis.h"
/*
 * X13 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014-2016  phm, Provos Alexis
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software", to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   phm <phm@inbox.com>
 * @author   Provos Alexis (Applied partial shared Mem utilization under CUDA 7.5 for compute5.0/5.2 / 2016)
 */

static __constant__ const uint32_t c_S[16] = {
		0x8807a57e, 0xe616af75, 0xc5d3e4db, 0xac9ab027,
		0xd915f117, 0xb6eecc54, 0x06e8020b, 0x4a92efd1,
		0xaac6e2c9, 0xddb21398, 0xcae65838, 0x437f203f,
		0x25ea78e7, 0x951fddd6, 0xda6ed11d, 0xe13e3567
};

static __device__ uint32_t mixtab0[256] = {
	0x63633297, 0x7c7c6feb, 0x77775ec7, 0x7b7b7af7, 0xf2f2e8e5, 0x6b6b0ab7,	0x6f6f16a7, 0xc5c56d39, 0x303090c0, 0x01010704, 0x67672e87, 0x2b2bd1ac, 0xfefeccd5, 0xd7d71371, 0xabab7c9a,
	0x767659c3, 0xcaca4005, 0x8282a33e, 0xc9c94909, 0x7d7d68ef, 0xfafad0c5,	0x5959947f, 0x4747ce07, 0xf0f0e6ed, 0xadad6e82, 0xd4d41a7d, 0xa2a243be, 0xafaf608a, 0x9c9cf946, 0xa4a451a6,
	0x727245d3, 0xc0c0762d, 0xb7b728ea, 0xfdfdc5d9, 0x9393d47a, 0x2626f298, 0x363682d8, 0x3f3fbdfc, 0xf7f7f3f1, 0xcccc521d, 0x34348cd0, 0xa5a556a2, 0xe5e58db9, 0xf1f1e1e9, 0x71714cdf,
	0xd8d83e4d, 0x313197c4, 0x15156b54, 0x04041c10, 0xc7c76331, 0x2323e98c, 0xc3c37f21, 0x18184860, 0x9696cf6e, 0x05051b14, 0x9a9aeb5e, 0x0707151c, 0x12127e48, 0x8080ad36, 0xe2e298a5,
	0xebeba781, 0x2727f59c, 0xb2b233fe, 0x757550cf, 0x09093f24, 0x8383a43a, 0x2c2cc4b0, 0x1a1a4668, 0x1b1b416c, 0x6e6e11a3, 0x5a5a9d73, 0xa0a04db6, 0x5252a553, 0x3b3ba1ec, 0xd6d61475,
	0xb3b334fa, 0x2929dfa4, 0xe3e39fa1, 0x2f2fcdbc, 0x8484b126, 0x5353a257, 0xd1d10169, 0x00000000, 0xededb599, 0x2020e080, 0xfcfcc2dd, 0xb1b13af2, 0x5b5b9a77, 0x6a6a0db3, 0xcbcb4701,
	0xbebe17ce, 0x3939afe4, 0x4a4aed33, 0x4c4cff2b, 0x5858937b, 0xcfcf5b11, 0xd0d0066d, 0xefefbb91, 0xaaaa7b9e, 0xfbfbd7c1, 0x4343d217, 0x4d4df82f, 0x333399cc, 0x8585b622, 0x4545c00f,
	0xf9f9d9c9, 0x02020e08, 0x7f7f66e7, 0x5050ab5b, 0x3c3cb4f0, 0x9f9ff04a, 0xa8a87596, 0x5151ac5f, 0xa3a344ba, 0x4040db1b, 0x8f8f800a, 0x9292d37e, 0x9d9dfe42, 0x3838a8e0, 0xf5f5fdf9,
	0xbcbc19c6, 0xb6b62fee, 0xdada3045, 0x2121e784, 0x10107040, 0xffffcbd1, 0xf3f3efe1, 0xd2d20865, 0xcdcd5519, 0x0c0c2430, 0x1313794c, 0xececb29d, 0x5f5f8667, 0x9797c86a, 0x4444c70b,
	0x1717655c, 0xc4c46a3d, 0xa7a758aa, 0x7e7e61e3, 0x3d3db3f4, 0x6464278b, 0x5d5d886f, 0x19194f64, 0x737342d7, 0x60603b9b, 0x8181aa32, 0x4f4ff627, 0xdcdc225d, 0x2222ee88, 0x2a2ad6a8,
	0x9090dd76, 0x88889516, 0x4646c903, 0xeeeebc95, 0xb8b805d6, 0x14146c50, 0xdede2c55, 0x5e5e8163, 0x0b0b312c, 0xdbdb3741, 0xe0e096ad, 0x32329ec8, 0x3a3aa6e8, 0x0a0a3628, 0x4949e43f,
	0x06061218, 0x2424fc90, 0x5c5c8f6b, 0xc2c27825, 0xd3d30f61, 0xacac6986, 0x62623593, 0x9191da72, 0x9595c662, 0xe4e48abd, 0x797974ff, 0xe7e783b1, 0xc8c84e0d, 0x373785dc, 0x6d6d18af,
	0x8d8d8e02, 0xd5d51d79, 0x4e4ef123, 0xa9a97292, 0x6c6c1fab, 0x5656b943, 0xf4f4fafd, 0xeaeaa085, 0x6565208f, 0x7a7a7df3, 0xaeae678e, 0x08083820, 0xbaba0bde, 0x787873fb, 0x2525fb94,
	0x2e2ecab8, 0x1c1c5470, 0xa6a65fae, 0xb4b421e6, 0xc6c66435, 0xe8e8ae8d, 0xdddd2559, 0x747457cb, 0x1f1f5d7c, 0x4b4bea37, 0xbdbd1ec2, 0x8b8b9c1a, 0x8a8a9b1e, 0x70704bdb, 0x3e3ebaf8,
	0xb5b526e2, 0x66662983, 0x4848e33b, 0x0303090c, 0xf6f6f4f5, 0x0e0e2a38, 0x61613c9f, 0x35358bd4, 0x5757be47, 0xb9b902d2, 0x8686bf2e, 0xc1c17129, 0x1d1d5374, 0x9e9ef74e, 0xe1e191a9,
	0xf8f8decd, 0x9898e556, 0x11117744, 0x696904bf, 0xd9d93949, 0x8e8e870e, 0x9494c166, 0x9b9bec5a, 0x1e1e5a78, 0x8787b82a, 0xe9e9a989, 0xcece5c15, 0x5555b04f, 0x2828d8a0, 0xdfdf2b51,
	0x8c8c8906, 0xa1a14ab2, 0x89899212, 0x0d0d2334, 0xbfbf10ca, 0xe6e684b5, 0x4242d513, 0x686803bb, 0x4141dc1f, 0x9999e252, 0x2d2dc3b4, 0x0f0f2d3c, 0xb0b03df6, 0x5454b74b, 0xbbbb0cda,
	0x16166258
};

#define mixtab0(x) shared[0][x]
#define mixtab1(x) shared[1][x]
#define mixtab2(x) shared[2][x]
#define mixtab3(x) shared[3][x]

#define TIX4(q, x00, x01, x04, x07, x08, x22, x24, x27, x30) { \
		x22 ^= x00; \
		x00 = (q); \
		x08 ^= (q); \
		x01 ^= x24; \
		x04 ^= x27; \
		x07 ^= x30; \
	}

#define CMIX36(x00, x01, x02, x04, x05, x06, x18, x19, x20) { \
		x00 ^= x04; \
		x01 ^= x05; \
		x02 ^= x06; \
		x18 ^= x04; \
		x19 ^= x05; \
		x20 ^= x06; \
	}

__device__ __forceinline__
static void SMIX(const uint32_t shared[4][256], uint32_t &x0,uint32_t &x1,uint32_t &x2,uint32_t &x3){
	uint32_t c0 = mixtab0(__byte_perm(x0,0,0x4443));
	uint32_t r1 = mixtab1(__byte_perm(x0,0,0x4442));
	uint32_t r2 = mixtab2(__byte_perm(x0,0,0x4441));
	uint32_t r3 = mixtab3(__byte_perm(x0,0,0x4440));
	c0 = c0 ^ r1 ^ r2 ^ r3;
	uint32_t r0 = mixtab0(__byte_perm(x1,0,0x4443));
	uint32_t c1 = r0 ^ mixtab1(__byte_perm(x1,0,0x4442));
	uint32_t tmp = mixtab2(__byte_perm(x1,0,0x4441));
	c1 ^= tmp;
	r2 ^= tmp;
	tmp = mixtab3(__byte_perm(x1,0,0x4440));
	c1 ^= tmp;
	r3 ^= tmp;
	uint32_t c2 = mixtab0(__byte_perm(x2,0,0x4443));
	r0 ^= c2;
	tmp = mixtab1(__byte_perm(x2,0,0x4442));
	c2 ^= tmp;
	r1 ^= tmp;
	tmp = mixtab2(__byte_perm(x2,0,0x4441));
	c2 ^= tmp;
	tmp = mixtab3(__byte_perm(x2,0,0x4440));
	c2 ^= tmp;
	r3 ^= tmp;
	uint32_t c3 = mixtab0(__byte_perm(x3,0,0x4443));
	r0 ^= c3;
	tmp = mixtab1(__byte_perm(x3,0,0x4442));
	c3 ^= tmp;
	r1 ^= tmp;
	tmp = mixtab2(__byte_perm(x3,0,0x4441));
	c3 ^= tmp;
	r2 ^= tmp;
	tmp = mixtab3(__byte_perm(x3,0,0x4440));
	c3 ^= tmp;
	x0 = ((c0 ^ (r0 << 0)) & 0xFF000000) | ((c1 ^ (r1 << 0)) & 0x00FF0000) | ((c2 ^ (r2 << 0)) & 0x0000FF00) | ((c3 ^ (r3 << 0)) & 0x000000FF);
	x1 = ((c1 ^ (r0 << 8)) & 0xFF000000) | ((c2 ^ (r1 << 8)) & 0x00FF0000) | ((c3 ^ (r2 << 8)) & 0x0000FF00) | ((c0 ^ (r3 >>24)) & 0x000000FF);
	x2 = ((c2 ^ (r0 <<16)) & 0xFF000000) | ((c3 ^ (r1 <<16)) & 0x00FF0000) | ((c0 ^ (r2 >>16)) & 0x0000FF00) | ((c1 ^ (r3 >>16)) & 0x000000FF);
	x3 = ((c3 ^ (r0 <<24)) & 0xFF000000) | ((c0 ^ (r1 >> 8)) & 0x00FF0000) | ((c1 ^ (r2 >> 8)) & 0x0000FF00) | ((c2 ^ (r3 >> 8)) & 0x000000FF);
}

__device__
static void SMIX_LDG(const uint32_t shared[4][256], uint32_t &x0,uint32_t &x1,uint32_t &x2,uint32_t &x3){
	uint32_t c0 = __ldg(&mixtab0[__byte_perm(x0,0,0x4443)]);
	uint32_t r1 = mixtab1(__byte_perm(x0,0,0x4442));
	uint32_t r2 = mixtab2(__byte_perm(x0,0,0x4441));
	uint32_t r3 = mixtab3(__byte_perm(x0,0,0x4440));
	c0 = c0 ^ r1 ^ r2 ^ r3;
	uint32_t r0 = __ldg(&mixtab0[__byte_perm(x1,0,0x4443)]);
	uint32_t c1 = r0 ^ mixtab1(__byte_perm(x1,0,0x4442));
	uint32_t tmp = mixtab2(__byte_perm(x1,0,0x4441));
	c1 ^= tmp;
	r2 ^= tmp;
	tmp = mixtab3(__byte_perm(x1,0,0x4440));
	c1 ^= tmp;
	r3 ^= tmp;
	uint32_t c2 = __ldg(&mixtab0[__byte_perm(x2,0,0x4443)]);
	r0 ^= c2;
	tmp = mixtab1(__byte_perm(x2,0,0x4442));
	c2 ^= tmp;
	r1 ^= tmp;
	tmp = mixtab2(__byte_perm(x2,0,0x4441));
	c2 ^= tmp;
	tmp = mixtab3(__byte_perm(x2,0,0x4440));
	c2 ^= tmp;
	r3 ^= tmp;
	uint32_t c3 = __ldg(&mixtab0[__byte_perm(x3,0,0x4443)]);
	r0 ^= c3;
	tmp = mixtab1(__byte_perm(x3,0,0x4442));
	c3 ^= tmp;
	r1 ^= tmp;
	tmp = mixtab2(__byte_perm(x3,0,0x4441));
	c3 ^= tmp;
	r2 ^= tmp;
	tmp = ROL8(__ldg(&mixtab0[__byte_perm(x3,0,0x4440)]));
	c3 ^= tmp;
	x0 = ((c0 ^ (r0 << 0)) & 0xFF000000) | ((c1 ^ (r1 << 0)) & 0x00FF0000) | ((c2 ^ (r2 << 0)) & 0x0000FF00) | ((c3 ^ (r3 << 0)) & 0x000000FF);
	x1 = ((c1 ^ (r0 << 8)) & 0xFF000000) | ((c2 ^ (r1 << 8)) & 0x00FF0000) | ((c3 ^ (r2 << 8)) & 0x0000FF00) | ((c0 ^ (r3 >>24)) & 0x000000FF);
	x2 = ((c2 ^ (r0 <<16)) & 0xFF000000) | ((c3 ^ (r1 <<16)) & 0x00FF0000) | ((c0 ^ (r2 >>16)) & 0x0000FF00) | ((c1 ^ (r3 >>16)) & 0x000000FF);
	x3 = ((c3 ^ (r0 <<24)) & 0xFF000000) | ((c0 ^ (r1 >> 8)) & 0x00FF0000) | ((c1 ^ (r2 >> 8)) & 0x0000FF00) | ((c2 ^ (r3 >> 8)) & 0x000000FF);
}
#define mROR3 { \
	B[ 6] = S[33], B[ 7] = S[34], B[ 8] = S[35]; \
	S[35] = S[32]; S[34] = S[31]; S[33] = S[30]; S[32] = S[29]; S[31] = S[28]; S[30] = S[27]; S[29] = S[26]; S[28] = S[25]; S[27] = S[24]; \
	S[26] = S[23]; S[25] = S[22]; S[24] = S[21]; S[23] = S[20]; S[22] = S[19]; S[21] = S[18]; S[20] = S[17]; S[19] = S[16]; S[18] = S[15]; \
	S[17] = S[14]; S[16] = S[13]; S[15] = S[12]; S[14] = S[11]; S[13] = S[10]; S[12] = S[ 9]; S[11] = S[ 8]; S[10] = S[ 7]; S[ 9] = S[ 6]; \
	S[ 8] = S[ 5]; S[ 7] = S[ 4]; S[ 6] = S[ 3]; S[ 5] = S[ 2]; S[ 4] = S[ 1]; S[ 3] = S[ 0]; S[ 2] = B[ 8]; S[ 1] = B[ 7]; S[ 0] = B[ 6]; \
	}

#define mROR8 { \
	B[ 1] = S[28], B[ 2] = S[29], B[ 3] = S[30], B[ 4] = S[31], B[ 5] = S[32], B[ 6] = S[33], B[ 7] = S[34], B[ 8] = S[35]; \
	S[35] = S[27]; S[34] = S[26]; S[33] = S[25]; S[32] = S[24]; S[31] = S[23]; S[30] = S[22]; S[29] = S[21]; S[28] = S[20]; S[27] = S[19]; \
	S[26] = S[18]; S[25] = S[17]; S[24] = S[16]; S[23] = S[15]; S[22] = S[14]; S[21] = S[13]; S[20] = S[12]; S[19] = S[11]; S[18] = S[10]; \
	S[17] = S[ 9]; S[16] = S[ 8]; S[15] = S[ 7]; S[14] = S[ 6]; S[13] = S[ 5]; S[12] = S[ 4]; S[11] = S[ 3]; S[10] = S[ 2]; S[ 9] = S[ 1]; \
	S[ 8] = S[ 0]; S[ 7] = B[ 8]; S[ 6] = B[ 7]; S[ 5] = B[ 6]; S[ 4] = B[ 5]; S[ 3] = B[ 4]; S[ 2] = B[ 3]; S[ 1] = B[ 2]; S[ 0] = B[ 1]; \
	}

#define mROR9 { \
	B[ 0] = S[27], B[ 1] = S[28], B[ 2] = S[29], B[ 3] = S[30], B[ 4] = S[31], B[ 5] = S[32], B[ 6] = S[33], B[ 7] = S[34], B[ 8] = S[35]; \
	S[35] = S[26]; S[34] = S[25]; S[33] = S[24]; S[32] = S[23]; S[31] = S[22]; S[30] = S[21]; S[29] = S[20]; S[28] = S[19]; S[27] = S[18]; \
	S[26] = S[17]; S[25] = S[16]; S[24] = S[15]; S[23] = S[14]; S[22] = S[13]; S[21] = S[12]; S[20] = S[11]; S[19] = S[10]; S[18] = S[ 9]; \
	S[17] = S[ 8]; S[16] = S[ 7]; S[15] = S[ 6]; S[14] = S[ 5]; S[13] = S[ 4]; S[12] = S[ 3]; S[11] = S[ 2]; S[10] = S[ 1]; S[ 9] = S[ 0]; \
	S[ 8] = B[ 8]; S[ 7] = B[ 7]; S[ 6] = B[ 6]; S[ 5] = B[ 5]; S[ 4] = B[ 4]; S[ 3] = B[ 3]; S[ 2] = B[ 2]; S[ 1] = B[ 1]; S[ 0] = B[ 0]; \
	}

#define FUGUE512_3(x, y, z) {  \
        TIX4(x, S[ 0], S[ 1], S[ 4], S[ 7], S[ 8], S[22], S[24], S[27], S[30]); \
        CMIX36(S[33], S[34], S[35], S[ 1], S[ 2], S[ 3], S[15], S[16], S[17]); \
        SMIX_LDG(shared, S[33], S[34], S[35], S[ 0]); \
        CMIX36(S[30], S[31], S[32], S[34], S[35], S[ 0], S[12], S[13], S[14]); \
        SMIX_LDG(shared, S[30], S[31], S[32], S[33]); \
        CMIX36(S[27], S[28], S[29], S[31], S[32], S[33], S[ 9], S[10], S[11]); \
        SMIX(shared, S[27], S[28], S[29], S[30]); \
        CMIX36(S[24], S[25], S[26], S[28], S[29], S[30], S[ 6], S[ 7], S[ 8]); \
        SMIX_LDG(shared, S[24], S[25], S[26], S[27]); \
        \
        TIX4(y, S[24], S[25], S[28], S[31], S[32], S[10], S[12], S[15], S[18]); \
        CMIX36(S[21], S[22], S[23], S[25], S[26], S[27], S[ 3], S[ 4], S[ 5]); \
        SMIX(shared, S[21], S[22], S[23], S[24]); \
        CMIX36(S[18], S[19], S[20], S[22], S[23], S[24], S[ 0], S[ 1], S[ 2]); \
        SMIX_LDG(shared, S[18], S[19], S[20], S[21]); \
        CMIX36(S[15], S[16], S[17], S[19], S[20], S[21], S[33], S[34], S[35]); \
        SMIX_LDG(shared, S[15], S[16], S[17], S[18]); \
        CMIX36(S[12], S[13], S[14], S[16], S[17], S[18], S[30], S[31], S[32]); \
        SMIX(shared, S[12], S[13], S[14], S[15]); \
        \
        TIX4(z, S[12], S[13], S[16], S[19], S[20], S[34], S[ 0], S[ 3], S[ 6]); \
        CMIX36(S[ 9], S[10], S[11], S[13], S[14], S[15], S[27], S[28], S[29]); \
        SMIX_LDG(shared, S[ 9], S[10], S[11], S[12]); \
        CMIX36(S[ 6], S[ 7], S[ 8], S[10], S[11], S[12], S[24], S[25], S[26]); \
        SMIX_LDG(shared, S[ 6], S[ 7], S[ 8], S[ 9]); \
        CMIX36(S[ 3], S[ 4], S[ 5], S[ 7], S[ 8], S[ 9], S[21], S[22], S[23]); \
        SMIX_LDG(shared, S[ 3], S[ 4], S[ 5], S[ 6]); \
        CMIX36(S[ 0], S[ 1], S[ 2], S[ 4], S[ 5], S[ 6], S[18], S[19], S[20]); \
        SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]); \
	}

/***************************************************/
// Die Hash-Funktion
__global__ __launch_bounds__(256,3)
void x13_fugue512_gpu_hash_64_alexis(uint32_t threads, uint64_t *g_hash)
{
	__shared__ uint32_t shared[4][256];

//	if(threadIdx.x<256){
		const uint32_t tmp = mixtab0[threadIdx.x];
		shared[0][threadIdx.x] = tmp;
		shared[1][threadIdx.x] = ROR8(tmp);
		shared[2][threadIdx.x] = ROL16(tmp);
		shared[3][threadIdx.x] = ROL8(tmp);
//	}
	__syncthreads();
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *hash = (uint32_t*)&g_hash[thread<<3];

		uint32_t S[36];
		uint32_t B[ 9];

		uint32_t Hash[16];

		*(uint2x4*)&Hash[0] = __ldg4((uint2x4*)&hash[0]);
		*(uint2x4*)&Hash[8] = __ldg4((uint2x4*)&hash[8]);

		#pragma unroll 16
		for(int i = 0; i < 16; i++)
			Hash[i] = cuda_swab32(Hash[i]);

		__syncthreads();

		S[ 0] = S[ 1] = S[ 2] = S[ 3] = S[ 4] = S[ 5] = S[ 6] = S[ 7] = S[ 8] = S[ 9] = S[10] = S[11] = S[12] = S[13] = S[14] = S[15] = S[16] = S[17] = S[18] = S[19] = 0;
		*(uint2x4*)&S[20] = *(uint2x4*)&c_S[ 0];
		*(uint2x4*)&S[28] = *(uint2x4*)&c_S[ 8];

		FUGUE512_3(Hash[0x0], Hash[0x1], Hash[0x2]);
		FUGUE512_3(Hash[0x3], Hash[0x4], Hash[0x5]);
		FUGUE512_3(Hash[0x6], Hash[0x7], Hash[0x8]);
		FUGUE512_3(Hash[0x9], Hash[0xA], Hash[0xB]);
		FUGUE512_3(Hash[0xC], Hash[0xD], Hash[0xE]);
		FUGUE512_3(Hash[0xF], 0U, 512U);

		for (uint32_t i = 0; i < 32; i+=2){
			mROR3;
			CMIX36(S[ 0], S[ 1], S[ 2], S[ 4], S[ 5], S[ 6], S[18], S[19], S[20]);
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			mROR3;
			CMIX36(S[ 0], S[ 1], S[ 2], S[ 4], S[ 5], S[ 6], S[18], S[19], S[20]);
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		}
		#pragma unroll
		for (uint32_t i = 0; i < 13; i ++) {
			S[ 4] ^= S[ 0];	S[ 9] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[19] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[19] ^= S[ 0];	S[28] ^= S[ 0];
			mROR8;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		}
		S[ 4] ^= S[ 0];	S[ 9] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];

		S[ 0] = cuda_swab32(S[ 1]);	S[ 1] = cuda_swab32(S[ 2]);	S[ 2] = cuda_swab32(S[ 3]);	S[ 3] = cuda_swab32(S[ 4]);
		S[ 4] = cuda_swab32(S[ 9]);	S[ 5] = cuda_swab32(S[10]);	S[ 6] = cuda_swab32(S[11]);	S[ 7] = cuda_swab32(S[12]);
		S[ 8] = cuda_swab32(S[18]);	S[ 9] = cuda_swab32(S[19]);	S[10] = cuda_swab32(S[20]);	S[11] = cuda_swab32(S[21]);
		S[12] = cuda_swab32(S[27]);	S[13] = cuda_swab32(S[28]);	S[14] = cuda_swab32(S[29]);	S[15] = cuda_swab32(S[30]);

		*(uint2x4*)&hash[ 0] = *(uint2x4*)&S[ 0];
		*(uint2x4*)&hash[ 8] = *(uint2x4*)&S[ 8];
	}
}

/***************************************************/
// The final hash function
__global__ __launch_bounds__(512,2) /* force 56 registers */
void x13_fugue512_gpu_hash_64_final_alexis(uint32_t threads,const uint32_t* __restrict__ g_hash,uint32_t* resNonce, const uint64_t target){

	__shared__ uint32_t shared[4][256];

	if(threadIdx.x<256){
		const uint32_t tmp = mixtab0[threadIdx.x];
		shared[0][threadIdx.x] = tmp;
		shared[1][threadIdx.x] = ROR8(tmp);
		shared[2][threadIdx.x] = ROL16(tmp);
		shared[3][threadIdx.x] = ROL8(tmp);
	}

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t* __restrict__ hash = &g_hash[thread<<4];

		uint32_t S[36];
		uint32_t B[ 9];
		uint32_t Hash[16];

		*(uint2x4*)&Hash[0] = __ldg4((uint2x4*)&hash[0]);
		*(uint2x4*)&Hash[8] = __ldg4((uint2x4*)&hash[8]);
		__syncthreads();
		S[ 0] = S[ 1] = S[ 2] = S[ 3] = S[ 4] = S[ 5] = S[ 6] = S[ 7] = S[ 8] = S[ 9] = S[10] = S[11] = S[12] = S[13] = S[14] = S[15] = S[16] = S[17] = S[18] = S[19] = 0;
		*(uint2x4*)&S[20] = *(uint2x4*)&c_S[ 0];
		*(uint2x4*)&S[28] = *(uint2x4*)&c_S[ 8];

		FUGUE512_3(Hash[0x0], Hash[0x1], Hash[0x2]);
		FUGUE512_3(Hash[0x3], Hash[0x4], Hash[0x5]);
		FUGUE512_3(Hash[0x6], Hash[0x7], Hash[0x8]);
		FUGUE512_3(Hash[0x9], Hash[0xA], Hash[0xB]);
		FUGUE512_3(Hash[0xC], Hash[0xD], Hash[0xE]);
		FUGUE512_3(Hash[0xF], 0, 512);

		for (int i = 0; i < 32; i++){
			mROR3;
			CMIX36(S[ 0], S[ 1], S[ 2], S[ 4], S[ 5], S[ 6], S[18], S[19], S[20]);
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		}
		#pragma unroll
		for (int i = 0; i < 12; i++) {
			S[ 4] ^= S[ 0];	S[ 9] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[19] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[19] ^= S[ 0];	S[28] ^= S[ 0];
			mROR8;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		}
		S[ 4] ^= S[ 0];	S[ 9] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
		mROR9;
		SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
		mROR9;
		SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[19] ^= S[ 0];	S[27] ^= S[ 0];
		mROR9;
		SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);

		S[ 3] = cuda_swab32(S[3]);	S[ 4] = cuda_swab32(S[4]^S[ 0]);

		const uint64_t check = *(uint64_t*)&S[ 3];
		if(check <= target){
			uint32_t tmp = atomicExch(&resNonce[0], thread);
			if (tmp != UINT32_MAX)
				resNonce[1] = tmp;
		}
	}
}

__host__
void x13_fugue512_cpu_hash_64_alexis(int thr_id, uint32_t threads, uint32_t *d_hash){

	const uint32_t threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x13_fugue512_gpu_hash_64_alexis<<<grid, block>>>(threads, (uint64_t*)d_hash);
}

__host__
void x13_fugue512_cpu_hash_64_final_alexis(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t *d_resNonce, const uint64_t target){

	const uint32_t threadsperblock = 512;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x13_fugue512_gpu_hash_64_final_alexis<<<grid, block>>>(threads, d_hash,d_resNonce,target);
}
